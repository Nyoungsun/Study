#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>


void printVector(int* c, int* a, int* b, int arraySize) {
    int i;
    printf("{");
    for (i = 0; i < arraySize; i++) {
        printf("%d", a[i]);
        if (i != arraySize - 1) {
            printf(", ");
        }
        else {
            printf("}");
        }
    }

    printf(" + ");
    printf("{");
    for (i = 0; i < arraySize; i++) {
        printf("%d", b[i]);
        if (i != arraySize - 1) {
            printf(", ");
        }
        else {
            printf("}");
        }
    }

    printf(" = ");
    printf("{");
    for (i = 0; i < arraySize; i++) {
        printf("%d", c[i]);
        if (i != arraySize - 1) {
            printf(", ");
        }
        else {
            printf("}");
        }
    }
    printf("\n");
}
void RandArray(int* a, int* b, int arraySize) {
    srand(time(NULL));
    for (int i = 0; i < arraySize; i++) {
        a[i] = rand() % 100 + 1;
        b[i] = rand() % 100 + 1;
    }
}
__global__ void addKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, int* a, int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

void main()
{

    /*int *arrA , *tempA;
    int *arrB, *tempB;
    int* result;
    int size;

    printf("Enter your vector size: ");
    scanf("%d", &size);

    arrA = (int*)malloc(sizeof(int) * size);
    arrB = (int*)malloc(sizeof(int) * size);

    tempA = (int*)malloc(sizeof(int) * size);
    tempB = (int*)malloc(sizeof(int) * size);

    result = (int*)malloc(sizeof(int) * size);

    int i = 0;
    printf("Enter your vector A: ");
    while (size) {
        scanf("%d", tempA[i]);
        i++;
        size--;
    }

    i = 0;

    printf("Enter your vector B: ");
    while (size) {
        scanf("%d", tempB[i]);
        i++;
        size--;
    }

    for (i = 0; i < size; i++) {
        arrA[i] = tempA[i];
        arrB[i] = tempB[i];
    }*/
    //init vectorsize & input Vector
    const int arraySize = 5;
    int a[arraySize] = { 0, };
    int b[arraySize] = { 0, };
    int c[arraySize] = { 0, };
    
    RandArray(a, b, arraySize);

    hipEvent_t start, stop;
    float time;
    float sp, tp, t1;

    printf("+++Vector Summation by Parallel Proccessor+++\n\n");
    //Time record Start
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    //print Vector
    printVector(c, a, b, arraySize);
    printf("time(tp) : %f\n",time);
    tp = (int)time*10000;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    printf("\n+++Vector Summation by Single Proccessor+++\n\n");
    //Time record Start
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    //Employ single Proccessor
    for (int i = 0; i < arraySize; i++) {
        c[i] = a[i] + b[i];
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    //print Vector
    printVector(c, a, b, arraySize);
    printf("time(t1) : %f\n", time);
    t1 = (int)time * 10000;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    sp = t1 / tp;
    printf("\n\nArraySize:%d\nSp=%d (::t1/tp)\n",arraySize, sp);
}


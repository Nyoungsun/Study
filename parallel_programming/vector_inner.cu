#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>


void printVector(int* c, int* a, int* b, int arraySize) {
    int i;
    printf("{");
    for (i = 0; i < arraySize; i++) {
        printf("%d", a[i]);
        if (i != arraySize - 1) {
            printf(", ");
        }
        else {
            printf("}");
        }
    }

    printf(" · ");
    printf("{");
    for (i = 0; i < arraySize; i++) {
        printf("%d", b[i]);
        if (i != arraySize - 1) {
            printf(", ");
        }
        else {
            printf("}");
        }
    }

    int result = 0;
    for (i = 0; i < arraySize; i++) {
        result += c[i];
    }
    printf(" = {%d}\n", result);
}


void RandArray(int* a, int* b, int arraySize) {
    srand(time(NULL));
    for (int i = 0; i < arraySize; i++) {
        a[i] = rand() % 100 + 1;
        b[i] = rand() % 100 + 1;
    }
}
__global__ void multiKernel(int* c, int* a, int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t multiWithCuda(int* c, int* a, int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    multiKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

void main()
{
    //init vectorsize & input Vector
    const int arraySize = 5;
    int a[arraySize] = { 0, };
    int b[arraySize] = { 0, };
    int c[arraySize] = { 0, };

    RandArray(a, b, arraySize);

    hipEvent_t start, stop;
    float time;
    float sp, tp, t1;

    printf("+++Vector Inner Product by Parallel Proccessor+++\n\n");
    //Time record Start
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    // Add vectors in parallel.
    hipError_t cudaStatus = multiWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "multiWithCuda failed!");
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    //print Vector
    printVector(c, a, b, arraySize);
    printf("time(tp) : %f\n", time);
    tp = (int)time * 10000;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    printf("\n+++Vector Inner Product by Single Proccessor+++\n\n");
    //Time record Start
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    //Employ single Proccessor
    for (int i = 0; i < arraySize; i++) {
        c[i] = a[i] * b[i];
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    //print Vector
    printVector(c, a, b, arraySize);
    printf("time(t1) : %f\n", time);
    t1 = (int)time * 10000;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    sp = t1 / tp;
    printf("\n\nArraySize:%d\nSp=%d (::t1/tp)\n", arraySize, sp);
}